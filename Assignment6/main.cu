#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <stdio.h>

#define BLOCK_SIZE 1024

__device__ float function(float x)
{
  return x * x;
}

__global__ void init(unsigned int seed, hiprandState_t* states)
{
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed, blockIdx.x, 0, &states[id]);
}

__global__ void mc(int* ret,
                     hiprandState_t* states,
                     float a,
                     float b,
                     float h,
                     float (*f)(float))
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  float x = hiprand_uniform(&states[i]) * (b-a) + a;
  float y = hiprand_uniform(&states[i]) * (h-0) + 0;
  if (function(x) >= y)
    atomicAdd(ret, 1);
}

int main()
{
  int* h_a;
  int* d_a;
  float a = 1.0;
  float b = 10.0;
  float h = 150.0;
  unsigned int grid = 1000;
  hipError_t err;
  
  hiprandState_t* states;
  unsigned int n = BLOCK_SIZE * grid;

  h_a = (int*)malloc(sizeof(int));
  *h_a = 0;

  hipMalloc((void**)&d_a, sizeof(int));
  hipMalloc((void**)&states, n * sizeof(hiprandState_t));

  hipMemcpy(d_a, h_a, sizeof(int), hipMemcpyHostToDevice);

  dim3 dimBlock(BLOCK_SIZE, 1, 1);
  dim3 dimGrid(grid, 1, 1);

  init<<<dimGrid, dimBlock>>>((unsigned int) time(NULL), states);

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    fprintf(stderr, "init : %s\n", hipGetErrorString(err));
    exit(-1);
  }
  mc<<<dimGrid, dimBlock>>>(d_a, states, a, b, h, &function);
  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    fprintf(stderr, "mc : %s\n", hipGetErrorString(err));
    exit(-1);
  }

  hipMemcpy(h_a, d_a, sizeof(int), hipMemcpyDeviceToHost);

  printf("return: %d\n", *h_a);
  float result = ((b - a) * h) * ((float)*h_a / (float)n);
  printf("answer: %f\n", result);
  hipFree(d_a);
}
