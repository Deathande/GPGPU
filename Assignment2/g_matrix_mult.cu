#include "hip/hip_runtime.h"
#include "g_matrix_mult.h"


__global__
void sg_mat_mult(double* m1, double* m2, double* m3, unsigned int size)
{
  __shared__ double* s_m1;
  __shared__ double* s_m2;
  int c_i = threadIdx.x;
  int c_j = threadIdx.y;
  double dot = 0;

  s_m1 = m1;
  s_m2 = m2;
  for (int i = 0; i < size; i++)
    dot += s_m1[c_i * size + i] * s_m2[i * size + c_j];
  m3[c_i*size + c_j] = dot;
}

__global__
void g_mat_mult(double* m1, double* m2, double* m3, unsigned int size)
{
  int c_i = threadIdx.x;
  int c_j = threadIdx.y;
  double dot = 0;

  for (int i = 0; i < size; i++)
    dot += m1[c_i * size + i] * m2[i * size + c_j];
  m3[c_i*size + c_j] = dot;
}

double* shared_matrix_mult(const double* m1,
                           const double* m2,
			   unsigned int size)
{
  double* c_m1;
  double* c_m2;
  double* c_m3;
  double* result;
  unsigned int num_bytes;

  num_bytes = size * size * sizeof(double);
  result = (double*)malloc(num_bytes);
  hipMalloc((void**) &c_m1, num_bytes);
  hipMalloc((void**) &c_m2, num_bytes);
  hipMalloc((void**) &c_m3, num_bytes);

  hipMemcpy(c_m1, m1, num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(c_m2, m2, num_bytes, hipMemcpyHostToDevice);

  sg_mat_mult<<<ceil(size * size / 1024), size*size>>>(c_m1, c_m2, c_m3, size);
  hipMemcpy(result, c_m3, num_bytes, hipMemcpyDeviceToHost);

  hipFree(c_m1);
  hipFree(c_m2);
  hipFree(c_m3);

  return result;
}

double* global_matrix_mult(const double* m1,
                           const double* m2,
			   unsigned int size)
{
  int blocks = ceil(size * size / 1024);
  double* c_m1;
  double* c_m2;
  double* c_m3;
  double* result;
  unsigned int num_bytes;

  num_bytes = size * size * sizeof(double);
  result = (double*)malloc(num_bytes);
  hipMalloc((void**) &c_m1, num_bytes);
  hipMalloc((void**) &c_m2, num_bytes);
  hipMalloc((void**) &c_m3, num_bytes);

  hipMemcpy(c_m1, m1, num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(c_m2, m2, num_bytes, hipMemcpyHostToDevice);

  g_mat_mult<<<ceil(size * size / 1024), size*size>>>(c_m1, c_m2, c_m3, size);
  hipMemcpy(result, c_m3, num_bytes, hipMemcpyDeviceToHost);

  hipFree(c_m1);
  hipFree(c_m2);
  hipFree(c_m3);

  return result;
}

