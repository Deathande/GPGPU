#include "hip/hip_runtime.h"
#include "g_matrix_mult.h"

__global__ void g_mat_mult(double* m1, double* m2, double* m3, unsigned int size)
{
  int c_i = threadIdx.x;
  int c_j = threadIdx.y;
  double dot = 0;
  for (int i = 0; i < size; i++)
    dot += m1[c_i * size + c_j];
  m3[c_i*size + c_j];
}

double* global_matrix_mult(const double* m1, const double* m2, unsigned int size)
{
  double* c_m1;
  double* c_m2;
  double* c_m3;
  double* result;
  unsigned int num_bytes;

  num_bytes = size * size * sizeof(float);
  result = (double*)malloc(num_bytes);
  hipMalloc((void**) &c_m1, num_bytes);
  hipMalloc((void**) &c_m2, num_bytes);
  hipMalloc((void**) &c_m3, num_bytes);

  hipMemcpy(c_m1, m1, num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(c_m2, m2, num_bytes, hipMemcpyHostToDevice);

  g_mat_mult<<<size, size>>>(c_m1, c_m2, c_m3, size);
  hipMemcpy(result, c_m3, num_bytes, hipMemcpyDeviceToHost);

  hipFree(c_m1);
  hipFree(c_m2);
  hipFree(c_m3);

  return result;
}

